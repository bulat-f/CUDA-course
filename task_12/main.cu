
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define BASE_TYPE float

__global__ void dot_produce(const BASE_TYPE *a, const BASE_TYPE *b, BASE_TYPE *result, const int N)
{
    extern __shared__ BASE_TYPE s[];

    int index = blockDim.x * blockIdx.x + threadIdx.x;

    s[threadIdx.x] = a[index] * b[index];
    __syncthreads();
    
    if (threadIdx.x == 0) {
        for (int i = 1; i < blockDim.x; i++)
            s[0] += s[i];
        result[blockIdx.x] = s[0];
    }
}

BASE_TYPE* gen_array(const int N)
{
    BASE_TYPE *a = new BASE_TYPE[N];

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
    }

    return a;
}

void print_vector(BASE_TYPE *a, const int N)
{
    for (int i = 0; i < N; i++)
        printf("%3.0f ", a[i]);

    printf("\n");
}

void cuda_init_array(BASE_TYPE **dev, const BASE_TYPE *host, const size_t size)
{
    hipError_t err;
    err = hipMalloc((void **)dev, size);
    if (err != hipSuccess)
        throw err;

    if (host != NULL)
    {
        err = hipMemcpy(*dev, host, size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
            throw err;
    }
}

void cuda_init_grid_and_block(dim3 *grid, dim3 *block, const int threadsPerBlock, const int N)
{
    *grid = dim3(1);
    *block = dim3(N);
    printf("Block %d %d %d\n", block->x, block->y, block->z);
    printf("Grid %d %d %d\n", grid->x, grid->y, grid->z);
}

int main()
{
    const int N = 10;
    const int threadsPerBlock = N;
    const size_t size = N * sizeof(BASE_TYPE);
    const size_t result_size = size / threadsPerBlock;
    hipError_t err;

    dim3 blockDim, gridDim;
    cuda_init_grid_and_block(&blockDim, &gridDim, threadsPerBlock, N);

    BASE_TYPE *host_a = gen_array(N), *host_b = gen_array(N);
    BASE_TYPE *dev_a, *dev_b, *dev_c;
    BASE_TYPE result;

    print_vector(host_a, N);
    print_vector(host_b, N);

    try
    {
        cuda_init_array(&dev_a, host_a, size);
        cuda_init_array(&dev_b, host_b, size);
        cuda_init_array(&dev_c, NULL, sizeof(BASE_TYPE));
    }
    catch (hipError_t err)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dot_produce<<<blockDim, gridDim, threadsPerBlock * sizeof(BASE_TYPE)>>>(dev_a, dev_b, dev_c, N);

    err = hipMemcpy(&result, dev_c, result_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("%4.2f\n", result);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    delete[] host_a;
    delete[] host_b;

    return 0;
}