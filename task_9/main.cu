
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define BASE_TYPE float

__global__ void add(BASE_TYPE *a, BASE_TYPE *b, BASE_TYPE *c, const int N)
{
    int ind = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    
    c[ind] = a[ind] + b[ind];
}

BASE_TYPE* gen_array(const int N)
{
    BASE_TYPE *a = new BASE_TYPE[N * N];

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            a[i * N + j] = i * N + j;
    }

    return a;
}

void print_array(BASE_TYPE *a, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%3.0f ", a[i *N + j]);

        printf("\n");
    }

    printf("\n");
}

void cuda_init_array(BASE_TYPE **dev, const BASE_TYPE *host, const size_t size)
{
    hipError_t err;
    err = hipMalloc((void **)dev, size);
    if (err != hipSuccess)
        throw err;

    if (host != NULL)
    {
        err = hipMemcpy(*dev, host, size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
            throw err;
    }
}

void cuda_init_grid_and_block(dim3 *grid, dim3 *block, const int N)
{
    *grid = dim3(1);
    *block = dim3(N, N, 1);
    printf("Block %d %d %d\n", block->x, block->y, block->z);
    printf("Grid %d %d %d\n", grid->x, grid->y, grid->z);
}

int main()
{
    const int N = 10;
    const size_t size = N * N * sizeof(BASE_TYPE);
    hipError_t err;

    dim3 threadsPerBlock, blocksPerGrid;
    cuda_init_grid_and_block(&blocksPerGrid, &threadsPerBlock, N);
    hipEvent_t start, stop;
    float h2d_cp_span, d2h_cp_span, k_span;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);

    BASE_TYPE *host_a = gen_array(N);
    BASE_TYPE *dev_a, *dev_b, *dev_c;

    if (host_a == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    print_array(host_a, N);

    hipEventRecord(start, 0);

    try
    {
        cuda_init_array(&dev_a, host_a, size);
        cuda_init_array(&dev_b, host_a, size);
        cuda_init_array(&dev_c, NULL, size);
    }
    catch (hipError_t err)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&h2d_cp_span, start, stop);

    add<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&k_span, start, stop);


    err = hipMemcpy(host_a, dev_c, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&d2h_cp_span, start, stop);

    printf("Copy form host to device time: %.2f milliseconds\n", h2d_cp_span);
    printf("Run kernel time: %.2f milliseconds\n", k_span);
    printf("Copy form device to host time: %.2f milliseconds\n", d2h_cp_span);

    print_array(host_a, N);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    delete[] host_a;

    return 0;
}