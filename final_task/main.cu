#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
// подключене библиотеки cuBLAS
#include <hipblas.h>

// макрос для работы с индексами в стиле FORTRAN
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main()
{
    const int N = 4;
    hipblasHandle_t handle;
    float *dev_x, *dev_A, *dev_b;
    int *dev_info;
    int *info;
    float *x, *A, *b;

    x = (float *)malloc (N * sizeof(*x));
    b = (float *)malloc (N * sizeof(*b));
    A = (float *)malloc (N * N * sizeof(*A));
    info = (int *)malloc (1 * sizeof(*info));

    // инициализация матрицы и вектора правой части
    int ind = 11;
    srand(time(NULL));
    for(int j = 0; j < N; j++)
    {
        for(int i = 0; i < N; i++)
        {
            { 
                A[IDX2C(i,j,N)]= (float) (rand() % 100);
            }
             A[IDX2C(i,i,N)] += 100;
        }
        b[j] = 1.0f;
    }

    // выделяем память на GPU соответствующего размера для каждой переменной
    hipMalloc((void**)&dev_x, N * sizeof(*x));
    hipMalloc((void**)&dev_b, N * sizeof(*x));
    hipMalloc((void**)&dev_A, N * N * sizeof(*A));

    hipMalloc((void**)&dev_info, N * sizeof(*A));

    // инициализируем контекст cuBLAS
    hipblasCreate(&handle);

    // копируем вектор и матрицу из CPU в GPU
    hipblasSetVector(N, sizeof(*b), b, 1, dev_b, 1);
    hipblasSetMatrix(N, N, sizeof(*A), A, N, dev_A, N);


    hipblasSgetrfBatched(handle, N, &dev_A, N, NULL, dev_info, 1);
    
    // решаем нижнюю треугольню матрицу
    hipblasStrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, dev_A, N, dev_b, 1);

    // копируем результат из GPU в CPU
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%3.0f ", A[IDX2C(i,j,N)]);
        }
        printf("\n");
    }
    hipblasGetVector(N, sizeof(*x), dev_b, 1, x, 1);
    hipblasGetMatrix(N, N, sizeof(*A), dev_A, N, A, N);
    hipblasGetVector(1, sizeof(*info), dev_info, 1, info, 1);

    printf("\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%3.2f ", A[IDX2C(i,j,N)]);
        }
        printf("\n");
    }

    printf("\n info = %d\n", info[0]);
    
    // освобождаем память в GPU
    hipFree (dev_x);
    hipFree (dev_b);
    hipFree (dev_A);

    // уничтожаем контекс cuBLAS
    hipblasDestroy(handle);

    // освобождаем память в CPU
    free(x);
    free(b);
    free(A);

    return EXIT_SUCCESS;
}
