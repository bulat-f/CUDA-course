
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

__global__ void cuda_add(int a, int b, int *result)
{
    *result = a + b;
}

int main()
{
    int a = 2, b = 3;
    int h_result, *d_result;
    hipError_t err = hipSuccess;

    err = hipMalloc((void **)&d_result, sizeof(int));
    if (err != hipSuccess)
        printf("fail\n");

    cuda_add<<<1, 1>>>(a, b, d_result);

    err = hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        printf("fail\n");

    hipFree(d_result);

    printf("a + b == %d\n", h_result);
    return 0;
}