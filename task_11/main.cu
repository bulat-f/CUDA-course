
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define BASE_TYPE float

__global__ void add(const BASE_TYPE *a, BASE_TYPE *a_t, const int N)
{
    int i = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    
    int j = N * (blockDim.x * blockIdx.x + threadIdx.x) + blockDim.y * blockIdx.y + threadIdx.y;
    
    a_t[i] = a[j];
}

BASE_TYPE* gen_array(const int N)
{
    BASE_TYPE *a = new BASE_TYPE[N * N];

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            a[i * N + j] = i * N + j;
    }

    return a;
}

void print_array(BASE_TYPE *a, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%3.0f ", a[i *N + j]);

        printf("\n");
    }

    printf("\n");
}

void cuda_init_array(BASE_TYPE **dev, const BASE_TYPE *host, const size_t size)
{
    hipError_t err;
    err = hipMalloc((void **)dev, size);
    if (err != hipSuccess)
        throw err;

    if (host != NULL)
    {
        err = hipMemcpy(*dev, host, size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
            throw err;
    }
}

void cuda_init_grid_and_block(dim3 *grid, dim3 *block, const int N)
{
    *grid = dim3(1);
    *block = dim3(N, N, 1);
    printf("Block %d %d %d\n", block->x, block->y, block->z);
    printf("Grid %d %d %d\n", grid->x, grid->y, grid->z);
}

int main()
{
    const int N = 10;
    const size_t size = N * N * sizeof(BASE_TYPE);
    hipError_t err;

    dim3 threadsPerBlock, blocksPerGrid;
    cuda_init_grid_and_block(&blocksPerGrid, &threadsPerBlock, N);
    hipEvent_t start, stop;
    float h2d_cp_span, d2h_cp_span, k_span;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);

    BASE_TYPE *host_a = gen_array(N);
    BASE_TYPE *dev_a, *dev_b;

    if (host_a == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    print_array(host_a, N);

    hipEventRecord(start, 0);

    try
    {
        cuda_init_array(&dev_a, host_a, size);
        cuda_init_array(&dev_b, NULL, size);
    }
    catch (hipError_t err)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&h2d_cp_span, start, stop);

    add<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&k_span, start, stop);


    err = hipMemcpy(host_a, dev_b, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&d2h_cp_span, start, stop);

    printf("Copy form host to device time: %.2f milliseconds\n", h2d_cp_span);
    printf("Run kernel time: %.2f milliseconds\n", k_span);
    printf("Copy form device to host time: %.2f milliseconds\n", d2h_cp_span);

    print_array(host_a, N);

    hipFree(dev_a);
    hipFree(dev_b);

    delete[] host_a;

    return 0;
}