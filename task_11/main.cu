
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define BASE_TYPE float

__global__ void mult(const BASE_TYPE *a, const BASE_TYPE *b, BASE_TYPE *c, const int N, const int M)
{
    int i = N * (blockDim.y * blockIdx.y + threadIdx.y);
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    
    BASE_TYPE sum = 0;
    
    for (int k = 0; k < N; k++)
    {
        sum += a[i + k] * b[k * M + j];
    }
    
    int id = M * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    
    c[id] = sum;
}

BASE_TYPE* gen_array(const int N)
{
    BASE_TYPE *a = new BASE_TYPE[N * N];

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            a[i * N + j] = i * N + j;
    }

    return a;
}

void print_array(BASE_TYPE *a, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%5.0f ", a[i *N + j]);

        printf("\n");
    }

    printf("\n");
}

void cuda_init_array(BASE_TYPE **dev, const BASE_TYPE *host, const size_t size)
{
    hipError_t err;
    err = hipMalloc((void **)dev, size);
    if (err != hipSuccess)
        throw err;

    if (host != NULL)
    {
        err = hipMemcpy(*dev, host, size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
            throw err;
    }
}

void cuda_init_grid_and_block(dim3 *grid, dim3 *block, const int N)
{
    *grid = dim3(1);
    *block = dim3(N, N, 1);
    printf("Block %d %d %d\n", block->x, block->y, block->z);
    printf("Grid %d %d %d\n", grid->x, grid->y, grid->z);
}

int main()
{
    const int N = 10;
    const size_t size = N * N * sizeof(BASE_TYPE);
    hipError_t err;

    dim3 threadsPerBlock, blocksPerGrid;
    cuda_init_grid_and_block(&blocksPerGrid, &threadsPerBlock, N);

    BASE_TYPE *host_a = gen_array(N), *host_b = gen_array(N);
    BASE_TYPE *dev_a, *dev_b, *dev_c;

    if (host_a == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    print_array(host_a, N);
    print_array(host_b, N);

    try
    {
        cuda_init_array(&dev_a, host_a, size);
        cuda_init_array(&dev_b, host_b, size);
        cuda_init_array(&dev_c, NULL, size);
    }
    catch (hipError_t err)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    mult<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, N, N);

    err = hipMemcpy(host_a, dev_c, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    print_array(host_a, N);

    hipFree(dev_a);
    hipFree(dev_b);

    delete[] host_a;

    return 0;
}