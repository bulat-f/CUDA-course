
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define BASE_TYPE float

__global__ void init(BASE_TYPE *a, const int N)
{
    int id = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    a[id] = id;
}

BASE_TYPE* gen_array(const int N)
{
    BASE_TYPE *a = new BASE_TYPE[N * N];

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            a[i * N + j] = i * N + j;
    }

    return a;
}

void print_array(BASE_TYPE *a, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%3.0f ", a[i *N + j]);

        printf("\n");
    }

    printf("\n");
}

void cuda_init_array(BASE_TYPE **dev, const BASE_TYPE *host, const size_t size)
{
    hipError_t err;
    err = hipMalloc((void **)dev, size);
    if (err != hipSuccess)
        throw err;

    if (host != NULL)
    {
        err = hipMemcpy(*dev, host, size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
            throw err;
    }
}

void cuda_init_grid_and_block(dim3 *grid, dim3 *block, const int N)
{
    *grid = dim3(1);
    *block = dim3(N, N, 1);
    printf("Block %d %d %d\n", block->x, block->y, block->z);
    printf("Grid %d %d %d\n", grid->x, grid->y, grid->z);
}

int main()
{
    const int N = 10;
    const size_t size = N * N * sizeof(BASE_TYPE);
    hipError_t err;

    dim3 threadsPerBlock, blocksPerGrid;
    cuda_init_grid_and_block(&blocksPerGrid, &threadsPerBlock, N);

    BASE_TYPE *host_a = gen_array(N);
    BASE_TYPE *dev_a;

    if (host_a == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    print_array(host_a, N);

    try
    {
        cuda_init_array(&dev_a, NULL, size);
    }
    catch (hipError_t err)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    init<<<blocksPerGrid, threadsPerBlock>>>(dev_a, N);

    err = hipMemcpy(host_a, dev_a, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    print_array(host_a, N);

    hipFree(dev_a);

    delete[] host_a;

    return 0;
}