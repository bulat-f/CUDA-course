
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define BASE_TYPE float

__global__ void init(BASE_TYPE *a, const int N)
{
    int ind = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    a[ind] = ind;
}

BASE_TYPE* gen_array(const int N)
{
    BASE_TYPE *a = new BASE_TYPE[N * N];

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            a[i * N + j] = i * N + j;
    }

    return a;
}

void print_array(BASE_TYPE *a, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%3.0f ", a[i *N + j]);

        printf("\n");
    }

    printf("\n");
}

void cuda_init_array(BASE_TYPE **dev, const BASE_TYPE *host, const size_t size)
{
    hipError_t err;
    err = hipMalloc((void **)dev, size);
    if (err != hipSuccess)
        throw err;

    if (host != NULL)
    {
        err = hipMemcpy(*dev, host, size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
            throw err;
    }
}

void cuda_init_grid_and_block(dim3 *grid, dim3 *block, const int N)
{
    *grid = dim3(N);
    *block = dim3(N, 1, 1);
    printf("Block %d %d %d\n", block->x, block->y, block->z);
    printf("Grid %d %d %d\n", grid->x, grid->y, grid->z);
}

int main()
{
    const int N = 10;
    const size_t size = N * N * sizeof(BASE_TYPE);
    hipError_t err;

    dim3 threadsPerBlock, blocksPerGrid;
    cuda_init_grid_and_block(&blocksPerGrid, &threadsPerBlock, N);
    hipEvent_t start, stop;
    float h2d_cp_span, d2h_cp_span, k_span;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);

    BASE_TYPE *host_a = gen_array(N);
    BASE_TYPE *dev_a;

    if (host_a == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    print_array(host_a, N);

    hipEventRecord(start, 0);

    try
    {
        cuda_init_array(&dev_a, NULL, size);
    }
    catch (hipError_t err)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&h2d_cp_span, start, stop);

    init<<<blocksPerGrid, threadsPerBlock>>>(dev_a, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&k_span, start, stop);


    err = hipMemcpy(host_a, dev_a, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&d2h_cp_span, start, stop);

    printf("Copy form host to device time: %.2f milliseconds\n", h2d_cp_span);
    printf("Run kernel time: %.2f milliseconds\n", k_span);
    printf("Copy form device to host time: %.2f milliseconds\n", d2h_cp_span);

    print_array(host_a, N);

    hipFree(dev_a);

    delete[] host_a;

    return 0;
}