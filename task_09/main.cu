
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define BASE_TYPE float

__global__ void add(BASE_TYPE *a, BASE_TYPE *b, BASE_TYPE *c, const int N)
{
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    c[i * N + j] = a[i * N + j] + b[i * N + j];
}

BASE_TYPE* gen_array(const int N)
{
    BASE_TYPE *a = new BASE_TYPE[N * N];

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            a[i * N + j] = i * N + j;
    }

    return a;
}

void print_array(BASE_TYPE *a, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%3.0f ", a[i *N + j]);

        printf("\n");
    }

    printf("\n");
}

void cuda_init_array(BASE_TYPE **dev, const BASE_TYPE *host, const size_t size)
{
    hipError_t err;
    err = hipMalloc((void **)dev, size);
    if (err != hipSuccess)
        throw err;

    if (host != NULL)
    {
        err = hipMemcpy(*dev, host, size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
            throw err;
    }
}

void cuda_init_grid_and_block(dim3 *grid, dim3 *block, const int N)
{
    *grid = dim3(N);
    *block = dim3(N, 1, 1);
    printf("Block %d %d %d\n", block->x, block->y, block->z);
    printf("Grid %d %d %d\n", grid->x, grid->y, grid->z);
}

int main()
{
    const int N = 10;
    const size_t size = N * N * sizeof(BASE_TYPE);
    hipError_t err;

    dim3 threadsPerBlock, blocksPerGrid;
    cuda_init_grid_and_block(&blocksPerGrid, &threadsPerBlock, N);
    
    BASE_TYPE *host_a = gen_array(N);
    BASE_TYPE *dev_a, *dev_b, *dev_c;

    if (host_a == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    print_array(host_a, N);

    try
    {
        cuda_init_array(&dev_a, host_a, size);
        cuda_init_array(&dev_b, host_a, size);
        cuda_init_array(&dev_c, NULL, size);
    }
    catch (hipError_t err)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    add<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, N);

    err = hipMemcpy(host_a, dev_c, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device (error code: %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    print_array(host_a, N);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    delete[] host_a;

    return 0;
}